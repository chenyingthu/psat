#include "hip/hip_runtime.h"
#include "./inc/Psat.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "sm_20_atomic_functions.h"
#include "sm_13_double_functions.h"
#define atomic_double
#define pi 3.141592653589793
void MyMemcpy(double *dst,double *src,int n,int HTD){
	if(HTD==1)
		hipMemcpy(dst,src,n*sizeof(double),hipMemcpyHostToDevice);
	else if(HTD==0)
		hipMemcpy(dst,src,n*sizeof(double),hipMemcpyDeviceToHost);
	else 
		printf("wrong in MyMemcpy\n");
}
__global__ void MyMemcpyD2D_kernel(double *dst,double *src,int n,double c){
	int tid=threadIdx.x;
	while(tid<n){
		dst[tid]=src[tid]*c;
		tid+=1024;
	}
}
void MyMemcpyD2D(double *dst,double *src,int n,double c){
	 MyMemcpyD2D_kernel<<<1,1024>>>(dst,src,n,c);
}
#ifdef atomic_double
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif
void Psat::formDAEX_gpu(){
	int k=0;
	for ( int i = 0; i < dae.n; i += 1 ) {
		dae.X[k++]=dae.x[i];
	}
	for ( int i = 0; i < bus.n; i += 1 ) {
		dae.X[k++]=dae.a[i];
	}
	for ( int i = 0; i < bus.n; i += 1 ) {
		dae.X[k++]=dae.V[i];
	}
}
void Psat::DAEXtox_gpu(){
	int k=0;
	for ( int i = 0; i < dae.n; i += 1 ) {
		dae.x[i]=dae.X[k++];
	}
	for ( int i = 0; i < bus.n; i += 1 ) {
		dae.a[i]=dae.X[k++];
	}
	for ( int i = 0; i < bus.n; i += 1 ) {
		dae.V[i]=dae.X[k++];
	}
}
void Psat::DAEXtox_2_gpu(double *x_in){
	int k=0;
	for ( int i = 0; i < dae.n; i += 1 ) {
		dae.x[i]=x_in[k++];
	}
	for ( int i = 0; i < bus.n; i += 1 ) {
		dae.a[i]=x_in[k++];
	}
	for ( int i = 0; i < bus.n; i += 1 ) {
		dae.V[i]=x_in[k++];
	}
}

void Psat::dyn_f_store_gpu(int iFlag){
	for ( int iStep = 0; iStep < simu.nSteps; iStep += 1 ) {
		int iStart=(iStep)*(dae.n+2*bus.n);
		int iEnd=(iStep+1)*(dae.n+2*bus.n);
		record.t[record.nhis]=simu.t_cur+(iStep+1)*simu.tStep;
		int k=0;
		for ( int j = iStart; j < iEnd; j += 1 ) {
			record.x[record.nhis*(dae.n+2*bus.n)+k++]=dae.X[j];
		}
		iStart=(iStep)*dae.n;
		iEnd=(iStep+1)*dae.n;
		k=0;
		for ( int j = iStart; j < iEnd; j += 1 ) {
			record.f[record.nhis*(dae.n)+k++]=dae.f[j];
		}
		record.nhis++;
	}
}
void Psat::dyn_f_integration_gpu(int iFlag){
	if (simu.multiSteps==1){
		simu.nSteps=settings.dyn_MulStep_nSteps;
		simu.multiSteps=2;
	}
	if(settings.dyn_isPredict==1)
		dyn_f_prediction_gpu(iFlag);
	if(iFlag==1){
		dae.X=solver_jfng_gpu(dae.X);
		debug((char*)"daeX",dae.n+2*bus.n,dae.X);
	}
}
void Psat::dyn_f_prediction_gpu(int iFlag){//to do multsteps,...
	int ord=settings.dyn_predict_model;
	double *x=new double [dae.n+2*bus.n]; 
	for(int i = 0;i<dae.n+2*bus.n;++i){
		x[i]=dae.X[i];
	}
	int isPre=0;
	if(ord==1){
		if(record.t[record.nhis-1]>fault.tFaultEnd+settings.dyn_tStep*3){
			isPre=1;
			for ( int i = 0; i < dae.n+2*bus.n; i += 1 ) {
				x[i]=2*record.x[i+(record.nhis-1)*(dae.n+2*bus.n)]-record.x[i+(record.nhis-2)*(dae.n+2*bus.n)];
			}
		}
	}
	else if(ord==2){
		if(record.t[record.nhis-1]>fault.tFaultEnd+settings.dyn_tStep*8){
			isPre=1;
			for ( int i = 0; i < dae.n+2*bus.n; i += 1 ) {
				x[i]=3*record.x[i+(record.nhis-1)*(dae.n+2*bus.n)]-3*record.x[i+(record.nhis-2)*(dae.n+2*bus.n)]+record.x[i+(record.nhis-3)*(dae.n+2*bus.n)];
			}
		}
	}
	else if(ord==3){
		if(record.t[record.nhis-1]>fault.tFaultEnd+settings.dyn_tStep*12){
			isPre=1;
			for ( int i = 0; i < dae.n+2*bus.n; i += 1 ) {
				x[i]=4*record.x[i+(record.nhis-1)*(dae.n+2*bus.n)]-6*record.x[i+(record.nhis-2)*(dae.n+2*bus.n)]+4*record.x[i+(record.nhis-3)*(dae.n+2*bus.n)]-record.x[i+(record.nhis-4)*(dae.n+2*bus.n)];
			}
		}
	}
	if(isPre==0){
		simu.nSteps=1;
		simu.multiSteps=0;
	}
	for(int i = 0;i<dae.n+2*bus.n;++i){
		dae.X[i]=x[i];
	}
	delete []x;
}
double * Psat::solver_jfng_gpu(double *x){
	int n=dae.n+2*bus.n;
	double *sol=new double [n];
	double *sol_dev;
	hipMalloc((void**)&sol_dev, n * sizeof(double));
	double gamma=0.9;
	// double ierr=0;
	int maxit=solver.jfng.newton.maxit;
	
	double rat;
	double stop_tol=solver.jfng.newton.tol;
	solver.jfng.gmres.tol=solver.jfng.gmres.tol_fixed;
	double *f0=new double [n];
	double *f0_dev;
	hipMalloc((void**)&f0_dev, n * sizeof(double));
	double *fold=new double [n];
	double *fold_dev;
	hipMalloc((void**)&f0_dev, n * sizeof(double));
	int itc=0;
	f0=dyn_f_dae_gpu(x,simu.t_next);
	hipMemcpy(f0_dev,f0,n*sizeof(double),hipMemcpyHostToDevice);
	double *f0_temp;
	hipMalloc((void**)&f0_temp, n * sizeof(double));
	hipMemcpy(f0_temp,f0_dev,n*sizeof(double),hipMemcpyDeviceToHost);
	double fnrm=norm(n,f0)/sqrt((double)n); 
	double fnrmo=1;
	simu.neval=0;
	while(fnrm > stop_tol&& itc < maxit){
		rat=fnrm/fnrmo;
		fnrmo=fnrm;
		itc=itc+1;
		if(solver.jfng.newton.method==1&&(solver.jfng.precond.inner==1||solver.jfng.precond.outer==1))
		{
			updatePreconditioner_gpu(1);
		}
		if(solver.jfng.newton.method==1){
			pre_gmres_gpu(f0,x,step);
			pre_gmres(f0,x,step);
		}
		for ( int i = 0; i < n; i += 1 ) {
			fold[i]=f0[i];
		}
		for ( int i = 0; i < n; i += 1 ) {
			x[i]+=step[i];
		}
		f0=dyn_f_dae_gpu(x,simu.t_next);
		fnrm=norm(n,f0)/sqrt((double)n); 
		rat=fnrm/fnrmo;
		if(solver.jfng.precond.outer==1 && solver.jfng.precond.outerStop == 2 && solver.update.num < solver.update.maxnum){
			int i=solver.update.num;
			for(int j=0;j<n;++j){
				solver.deltx[j+i*n]=step[j];
				solver.delty[j+i*n]=f0[j]-fold[j];
			}
			solver.update.num++;
		}
		if(solver.jfng.gmres.tol>0){
			double etaold=solver.jfng.gmres.tol;
			double etanew=gamma*rat*rat;
			if(gamma*etaold*etaold>0.1)
				etanew=max(etanew,gamma*etaold*etaold);
			solver.jfng.gmres.tol=min(etanew,solver.jfng.gmres.tol);
			solver.jfng.gmres.tol=max(solver.jfng.gmres.tol,0.5*stop_tol/fnrm);
		}
	}
	for ( int i = 0; i < n; i += 1 ) {
		sol[i]=x[i];
	}
	delete []fold;
	delete []f0;
	return sol;
}
double * Psat::dyn_f_dae_gpu(double *x_in,double t0)//todo multiSteps
{
	int pos=-1;
	int n=dae.n+2*bus.n;
	double *x_rec=new double [n];
	double *f_rec=new double [dae.n];
	double *f_out=new double [n];
	DAEXtox_2_gpu(x_in);
	if(settings.dyn_lf!=1){
		for ( int i = 0; i < record.nhis; i += 1 ) {
			if(abs(record.t[i]-t0)<1e-8)
				pos=i;
		}
		if(pos==-1){
			printf("no such time moment in his record");
			return NULL;
		}
		for ( int i = 0; i < n; i += 1 ) {
			x_rec[i]=record.x[i+pos*n];
			if(i<dae.n){
				f_rec[i]=record.f[i+pos*dae.n];
			}
		}
	}
	// for ( int i = 0; i < bus.n; i += 1 ) {
	//   printf("daeV\t%.16lf\n",dae.V[i]);
	//   printf("daeV\t%.16lf\n",dae.X[i+dae.n+bus.n]);
	// }
	// getchar();
	fm_lf_1();
	fm_mn_1();
	fm_syn(1);
	fm_sw_1();
	for ( int i = 0; i < bus.n; i += 1 ) {
		dae.g[i]=dae.gp[i];
		dae.g[i+bus.n]=dae.gq[i];
	}
	for ( int i = 0; i < dae.n; i += 1 ) {
		dae.f[i]=0;
	}
	fm_syn(3);
	for(int i=0;i<n;++i)
		f_out[i]=0;
	if(settings.dyn_lf!=1){
		for ( int i = 0; i < dae.n; i += 1 ) {
			f_out[i]=x_in[i]-x_rec[i]-simu.tStep*0.5*(dae.f[i]+f_rec[i]);
		}
	}
	for ( int i = dae.n; i < n; i += 1 ) {
		f_out[i]=dae.g[i-dae.n];
	}
	delete []x_rec;
	delete []f_rec;
	return f_out;
}
__global__ void updatePreconditioner_kernel1(int n,int j,double dest,double *p,double *q,double *deltx,double *delty){
	int tid=threadIdx.x;
	
	while(tid<n){
		p[tid]=dest*deltx[tid+j*n];
		q[tid]=delty[tid+j*n];
		tid+=1024;
	}
	__syncthreads();
}
__global__ void updatePreconditioner_kernel2(int n,int j,double dest,double *p,double *q,double *deltx,double *delty){
	__syncthreads();
#ifdef atomic_double
	__shared__  double temp[1];
	double a;
#else
	__shared__  float temp[1];
	float a;
#endif
	temp[0]=0;
	__syncthreads();
	int tid=threadIdx.x;

	while(tid<n){
		a=deltx[tid+j*n]*q[tid];
		atomicAdd( &temp[0],a);
		tid+=1024;
	}
	
	__syncthreads();
	//printf("%d\n",count);
	//printf("%d\n",count);
	tid=threadIdx.x;
	//if(tid==0)
	//	printf("0 %f\n",temp);
	//else if(tid==1)
	//	printf("1 %f\n",temp);
	while(tid<n){
		q[tid]+=temp[0]*delty[tid+j*n];
		tid+=1024;
	}
	__syncthreads();
}
__global__ void updatePreconditioner_kernel3(int n,int j,double dest,double *p,double *q,double *deltx,double *delty){

#ifdef atomic_double
	__shared__  double temp[1];
	double a;
#else
	__shared__  float temp[1];
	float a;
#endif
	temp[0]=0;
	__syncthreads();
	int tid=threadIdx.x;
	//printf("%d %d\n",j,tid);
	while(tid<n){
		a=p[tid]*q[tid];
		atomicAdd( &temp[0],a);
		tid+=1024;
	}
	__syncthreads();
	////__syncthreads();
	////printf("temp\t%f\n",temp);
	tid=threadIdx.x;
	
	//printf("%d\n",tid);
	while(tid<n){
		deltx[tid+j*n]=p[tid]/temp[0];
		delty[tid+j*n]=p[tid]-q[tid];
		//printf("%lf\n",deltx[tid+j*n]);
		tid+=1024;
	}
	__syncthreads();
}
void Psat::updatePreconditioner_gpu(int iFlag){
	int n=dae.n+2*bus.n;
	double *p=new double[n]; 
	double *q=new double [n];
	double *p_dev;
	double *q_dev;
	hipMalloc((void**)&p_dev,n* sizeof(double));
	hipMalloc((void**)&q_dev,n* sizeof(double));
	hipMemset(p_dev,0,n*sizeof(double));
	hipMemset(q_dev,0,n*sizeof(double));
	double temp=0;
	hipMemcpy(solver.deltx_dev,solver.deltx,n*solver.update.maxnum * sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(solver.delty_dev,solver.delty,n*solver.update.maxnum * sizeof(double),hipMemcpyHostToDevice);
	while(solver.update.last_num+1<solver.update.num){
		updatePreconditioner_kernel1<<<1,n>>>(n,solver.update.last_num,solver.update.dest,p_dev,q_dev,solver.deltx_dev,solver.delty_dev);
		for ( int j = 0; j < solver.update.last_num; j += 1 ) {
			updatePreconditioner_kernel2<<<1,n>>>(n,j,solver.update.dest,p_dev,q_dev,solver.deltx_dev,solver.delty_dev);
		}
		updatePreconditioner_kernel3<<<1,n>>>(n,solver.update.last_num,solver.update.dest,p_dev,q_dev,solver.deltx_dev,solver.delty_dev);
		solver.update.last_num++;
	}//end of while
	hipMemcpy(solver.deltx,solver.deltx_dev,n*solver.update.maxnum * sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(solver.delty,solver.delty_dev,n*solver.update.maxnum * sizeof(double),hipMemcpyDeviceToHost);
	delete []p; 
	delete []q;
}
__global__ void pre_gmres_kernel1(double *g,double rho){
	int tid=threadIdx.x;
	if(tid==0)
		g[tid]=rho;
}
void Psat::pre_gmres_gpu(double *f0,double *xc,double *x){
	double errtol=solver.jfng.gmres.tol;
	int kmax=solver.jfng.gmres.maxit;
	int reorth=solver.jfng.gmres.reorth;
	int n=(dae.n+2*bus.n)*simu.nSteps;
	double rho;
	double *xc_dev;
	hipMalloc((void**)&xc_dev,n* sizeof(double));
	double *x_dev;
	hipMalloc((void**)&x_dev,n* sizeof(double));
	double *f0_dev;
	hipMalloc((void**)&x_dev,n* sizeof(double));
	double *r_dev;
	hipMalloc((void**)&r_dev,n* sizeof(double));
	double *z_dev;
	hipMalloc((void**)&z_dev,n*kmax* sizeof(double));
	double *v_dev;
	hipMalloc((void**)&v_dev,n*kmax* sizeof(double));
	double *h_dev;
	hipMalloc((void**)&h_dev,kmax*kmax* sizeof(double));
	double *g_dev;
	hipMalloc((void**)&g_dev,(kmax+1)* sizeof(double));
	double *s_dev;
	hipMalloc((void**)&s_dev,(kmax+1)* sizeof(double));
	double *c_dev;
	hipMalloc((void**)&c_dev,(kmax+1)* sizeof(double));
	hipMemset(g_dev,0,(kmax+1)* sizeof(double));
	hipMemset(s_dev,0,(kmax+1)* sizeof(double));
	hipMemset(c_dev,0,(kmax+1)* sizeof(double));
	hipMemset(x_dev,0,n* sizeof(double));
	double *r=new double [n];
	//double *test=new double [kmax+1];
	double *testn=new double [n];
	MyMemcpy(r_dev,f0,n,1);
	MyMemcpyD2D(r_dev,r_dev,n,-1.0);
	MyMemcpy(r,r_dev,n,0);
	
	rho=norm(n,r);
	//hipMemset(g_dev,1,(kmax+1)*sizeof(double));
	pre_gmres_kernel1<<<1,1>>>(g_dev,rho);
	//MyMemcpy(test,g_dev,kmax+1,0);
	//debug("g",kmax+1,test);
	errtol=errtol*rho;
	errstep=rho;
	if(rho<errtol)
		return;
	MyMemcpyD2D(v_dev,r_dev,n,1/rho);
	//MyMemcpy(testn,v_dev+n-1,1,0);
	//debug("test n",1,testn);
	int k=0;
	//while(0){
	while((rho > errtol)&&(k<kmax)){
		k++;
		int kk=k-1;
		MyMemcpyD2D(z_dev+kk*n,v_dev+kk*n,n,1.0);
		if(solver.jfng.precond.inner==1||solver.jfng.precond.outer==1){
			precondition_gpu(z_dev+kk*n);
		}
		//MyMemcpy(testn,z_dev+kk*n,n,0);
		//debug("z_dev",n,testn);
	}
//	while((rho > errtol)&&(k<kmax)){
//		for ( int i = 0; i < n; i += 1 ) {
//			z_temp[i]=solver.jfng.gmres.z[i+kk*n];
//		}
//		dirder(xc,z_temp,f0,v_temp);
//		if(solver.jfng.precond.inner==1 && solver.jfng.precond.innerStop == 2 && solver.update.num < solver.update.maxnum){
//			int i=solver.update.num;
//			for(int j=0;j<n;++j){
//				solver.deltx[j+i*n]=solver.jfng.gmres.z[j+kk*n];
//				solver.delty[j+i*n]=v_temp[j];
//			}
//			solver.update.num++;
//		}
//		double normav=norm(n,v_temp);
//
//		for ( int j = 0; j < k; j += 1 ) {
//			double temp=0;
//			for (int i=0;i<n;++i)
//				temp+=solver.jfng.gmres.v[i+j*n]*v_temp[i];
//			solver.jfng.gmres.h[kk+j*kmax]=temp;
//			for (int i=0;i<n;++i)
//				v_temp[i]=v_temp[i]-temp*solver.jfng.gmres.v[i+j*n];
//		}
//		solver.jfng.gmres.h[kk+(kk+1)*kmax]=norm(n,v_temp);
//		double normav2=solver.jfng.gmres.h[kk+(kk+1)*kmax];
//		if((reorth==1&&abs(normav+0.001*normav2-normav)<1e-8)||reorth==3)
//		{
//			for ( int j = 0; j < k; j += 1 ) {
//				double hr=0;
//				for (int i=0;i<n;++i)
//					hr+=solver.jfng.gmres.v[i+j*n]*v_temp[i];
//				solver.jfng.gmres.h[kk+j*kmax]+=hr;
//				for (int i=0;i<n;++i)
//					v_temp[i]=v_temp[i]-hr*solver.jfng.gmres.v[i+j*n];
//			}
//			solver.jfng.gmres.h[kk+(kk+1)*kmax]=norm(n,v_temp);
//		}
//		if(solver.jfng.gmres.h[kk+(kk+1)*kmax]!=0){
//			for ( int i = 0; i < n; i += 1 ) {
//				v_temp[i]=v_temp[i]/solver.jfng.gmres.h[kk+(kk+1)*kmax];
//			}
//		}
//		if(k>1)
//			givapp(solver.jfng.gmres.c,solver.jfng.gmres.s,solver.jfng.gmres.h,k-1);
//		double nu=sqrt(solver.jfng.gmres.h[kk+(kk+1)*kmax]*solver.jfng.gmres.h[kk+(kk+1)*kmax]+solver.jfng.gmres.h[kk+(kk)*kmax]*solver.jfng.gmres.h[kk+(kk)*kmax]);
//		if(nu!=0)
//		{
//			solver.jfng.gmres.c[kk]=solver.jfng.gmres.h[kk+kk*kmax]/nu;
//			solver.jfng.gmres.s[kk]=-solver.jfng.gmres.h[kk+(kk+1)*kmax]/nu;
//			solver.jfng.gmres.h[kk+kk*kmax]=solver.jfng.gmres.c[kk]*solver.jfng.gmres.h[kk+kk*kmax]-solver.jfng.gmres.s[kk]*solver.jfng.gmres.h[kk+(kk+1)*kmax];
//			solver.jfng.gmres.h[kk+(kk+1)*kmax]=0;
//			double w1=solver.jfng.gmres.c[kk]*g[kk]-solver.jfng.gmres.s[kk]*g[kk+1];
//			double w2=solver.jfng.gmres.s[kk]*g[kk]+solver.jfng.gmres.c[kk]*g[kk+1];
//			g[kk]=w1;
//			g[kk+1]=w2;
//		}
//		rho=abs(g[kk+1]);
//		for (int i=0;i<n;++i)
//			solver.jfng.gmres.v[i+(kk+1)*n]=v_temp[i];
//	}
//	double *h_temp=new double [k*k];
//	double *z_temp2=new double [n*k];
//	int *ipiv=new int[k];
//	for ( int i = 0; i < k; i += 1 ) {
//		for ( int j = 0; j < k; j += 1 ) {
//			h_temp[i+j*k]=solver.jfng.gmres.h[j+i*kmax];
//		}
//	}
//	for ( int i = 0; i < k; i += 1 ) {
//		for ( int j = 0; j < n; j += 1 ) {
//			z_temp2[j+i*n]=solver.jfng.gmres.z[j+i*n];
//		}
//	}
//	double alpha=1;
//	double beta=0;
//	
//#ifndef gpu
//	MyDgesv(0,k,1,h_temp,k,ipiv,g,k);
//#else
//	MyDeviceDgesv(0,k,1,h_temp,k,ipiv,g,k);
//#endif
//#ifndef gpu
//	MyDgemv(0,n,k,alpha,z_temp2,n,g,1,beta,x,1);
//#else
//	MyDeviceDgemv(0,n,k,alpha,z_temp2,n,g,1,beta,x,1);
//#endif
//	// debug("x",n,x);
//	inner_it_count=k; 
//	simu.neval+=k;
//	delete []r;
//	delete []g;
//	delete []v_temp;
//	delete []z_temp;
//	delete []h_temp;
//	delete []ipiv;
//	delete []z_temp2;
} 
void Psat::givapp_gpu(double *c,double *s,double *vin,int k){
	int kmax=solver.jfng.gmres.maxit;
	for ( int i = 0; i < k; i += 1 ) {
		double w1=c[i]*vin[k+i*kmax]-s[i]*vin[k+(i+1)*kmax];
		double w2=s[i]*vin[k+i*kmax]+c[i]*vin[k+(i+1)*kmax];
		vin[k+i*kmax]=w1;
		vin[k+(i+1)*kmax]=w2;
		// printf("c\t%lf\ts\t%lf\n",c[i],s[i]);
		// printf("w1\t%lf\tw2\t%lf\n",w1,w2);
	}
}
__global__ void precondition_kernel(int n,int j,double *x,double *deltx,double *delty){

#ifdef atomic_double
	__shared__  double temp[1];
	double a;
#else
	__shared__  float temp[1];
	float a;
#endif
	int tid=threadIdx.x;
	temp[0]=0;
	__syncthreads();
	while(tid<n){
		a=deltx[tid+j*n]*x[tid];
		atomicAdd( &temp[0],a);
		tid+=1024;
	}
	__syncthreads();
	tid=threadIdx.x;
	while(tid<n){
		x[tid]+=temp[0]*delty[tid+j*n];
		tid+=1024;
	}
	__syncthreads();
}
void Psat::precondition_gpu(double *x){//todo multiSteps
	int kmax=solver.jfng.gmres.maxit;
	int n=dae.n+2*bus.n;
	MyMemcpyD2D(x,x,n,(double)solver.update.dest);
	for ( int i = 0; i <solver.update.last_num; i += 1 ) {
		precondition_kernel<<<1,n>>>(n,i,x,solver.deltx_dev,solver.delty_dev);
	}
}
void Psat::dirder_gpu(double *x,double *w,double *f0,double *z){
	int n=dae.n+2*bus.n;
	double temp=norm(n,w);
	double epsnew=solver.jfng.findiff;
	if(temp<1e-10)
	{
		for ( int i = 0; i < n; i += 1 ) {
			z[i]=0;
		}
	}
	epsnew=epsnew/temp;
	temp=norm(n,x);
	if(temp>0)
		epsnew=epsnew*temp;
	double *del=new double [n];
	for ( int i = 0; i < n; i += 1 ) {
		del[i]=x[i]+epsnew*w[i];
	}
	double *f1=new double [n];
	f1=dyn_f_dae(del,simu.t_next);
	for ( int i = 0; i < n; i += 1 ) {
		z[i]=(f1[i]-f0[i])/epsnew;
	}
	delete []del;
	delete []f1;
}
void Psat::dyn_f_increaseTimeSteps_gpu(int iFlag){
	switch(simu.converged){
	case 1:
		if(simu.newton_iteration>=15)
			simu.tStep=max(simu.tStep*0.9,settings.dyn_tStep_min);
		if(simu.newton_iteration<=10)
			simu.tStep=min(simu.tStep*1.3,settings.dyn_tStep_max);
		if(settings.fixt)
			simu.tStep=settings.dyn_tStep;
		break;
	case 0:
		simu.tStep=settings.dyn_tStep*0.5;
		if(simu.tStep<settings.dyn_tStep_min)
			simu.tStep=settings.dyn_tStep_min;
		break;
	}
	simu.t_cur=simu.t_next;
	simu.t_next=simu.t_cur+simu.nSteps*simu.tStep;
	double tempo_min=simu.t_next;
	for ( int i = 0; i < 4*fault.n; i += 1 ) {
		if(simu.t_switch[i]-simu.t_cur>1e-6&&simu.t_next-simu.t_switch[i]>1e-6)
			if(tempo_min>simu.t_switch[i])
				tempo_min=simu.t_switch[i];
	}
	simu.t_next=tempo_min;
	simu.tStep=simu.t_next-simu.t_cur;
	// for ( int i = 0; i < 4*fault.n; i += 1 ) {
	//   if(simu.t_next<simu.t_switch[4*fault.n-1]&&simu.t_switch[i]<simu.t_next){
	//     printf("I am here!!!\n");
	//     printf("%lf\n%lf\n",simu.t_next,simu.t_switch[i]);
	//     simu.t_next=simu.t_switch[i];
	//     simu.tStep=simu.t_next-simu.t_cur;
	//     simu.t_switch[i]=simu.t_switch[4*fault.n-1]*2;
	//     break;
	//   }
	// }
	if(simu.t_next>fault.tFaultStart+simu.tStep){
		solver.jfng.precond.inner=1;
		solver.jfng.precond.outer=1;
		if(simu.neval>0&&simu.neval<8){
			solver.jfng.precond.innerStop=1;
			solver.jfng.precond.outerStop=1;
			if(settings.dyn_isMulStep==1)
				if(simu.multiSteps==0)
					simu.multiSteps=1;
		}
		else if (simu.neval>7){
			solver.jfng.precond.innerStop=2;
			solver.jfng.precond.outerStop=2;
		}
	}
}
void Psat::dyn_f_dealFaults_gpu(int iFalg){
	formDAEX_gpu();
	for ( int i = 0; i < fault.n; i += 1 ) {
		int h=fault.bus[i];
		if(abs(simu.t_next-fault.con[i][4])<1e-8){
			for ( int j = 0; j < bus.n; j += 1 ) {
				V_bak[j]=dae.V[j];
			}
			printf("applying fault at t = %lf s\n",simu.t_next);
			for (int j=0;j<bus.n;++j){

				dae.V[j]=0.6;
			}
			dae.V[h]=0.01;
			for(int j=0;j<syn.n;++j){
				int k=syn.bus[j];
				dae.V[k]=dae.X[dae.n+k+bus.n];
			}
			shunt.g[h]=fault.dat[2+i*5]+fault.dat[0+i*5];
			shunt.b[h]=fault.dat[3+i*5]+fault.dat[1+i*5];
			formDAEX_gpu();
			fm_y();
			dyn_f_iniSolver(2);
			settings.dyn_lf=1;
			dae.X=solver_jfng_gpu(dae.X);
			settings.dyn_lf=2;
		}// fault intervention
		else if(abs(simu.t_next-fault.con[i][5])<1e-8){
			printf("Clearing fault at t = %lf s\n",simu.t_next);
			//getchar();
			shunt.g[h]=fault.dat[2+i*5];
			shunt.b[h]=fault.dat[3+i*5];
			fm_y();
			printf("LY done\n");
			dyn_f_iniSolver(2);
			solver.update.dest=1;
			settings.dyn_lf=1;
			for ( int i = 0; i < bus.n; i += 1 ) {
				dae.V[i]=V_bak[i];
			}
			if(syn.n>0){
				double mean_delta=0;
				for ( int i = 0; i < syn.n; i += 1 ) {
					int k=syn.delta_idx[i];
					mean_delta+=dae.x[k];
				}
				mean_delta=mean_delta/syn.n;
				for ( int i = 0; i < bus.n-boundarynode.n; i += 1 ) {
					int k=boundarynode.indexG[i];
					dae.a[k]=mean_delta-fault.delta+fault.ang[k];
				}
			}
			else{
				for ( int i = 0; i < bus.n; i += 1 ) {
					dae.a[i]=fault.ang[i];
				}
			}
			formDAEX_gpu();
			dae.X=solver_jfng_gpu(dae.X);
			settings.dyn_lf=2;
		}//end of else if

	}//end of for
	formDAEX_gpu();
}
